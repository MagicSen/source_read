
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

__global__ void sync_conv_groups_() { }

//using std::cout;
//using std::endl;

template <typename Dtype>
void CuDNNConvolutionMaskLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    size_t workspace_limit_bytes = this->kernel_h_ *
                                   this->kernel_w_ *
                                   this->channels_ *
                                   sizeof(int) + 1;

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      cudnnConvolutionFwdAlgo_t algo;

      // pick the convolution algorithm
      // TODO(shelhamer) this should be done during reshape
      // TODO(shelhamer) the choice of automatic or manual algorithm picking
      // should be exposed in proto
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_limit_bytes,  // memoryLimitInBytes,
        &algo));

      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes_temp = 0;

      CUDNN_CHECK(cudnnGetConvolutionForwardWorkspaceSize(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        algo,
        &workspaceSizeInBytes_temp));

      if (workspaceSizeInBytes_temp > workspaceSizeInBytes) {
        workspaceSizeInBytes = workspaceSizeInBytes_temp;
        // free the existing workspace and allocate a new (larger) one
        cudaFree(this->workspace);
        cudaError_t err = cudaMalloc(&(this->workspace), workspaceSizeInBytes);
        if (err != cudaSuccess) {
          // force zero memory path
          algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
          workspace = NULL;
          workspaceSizeInBytes = 0;
        }
      }

      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + weight_offset_ * g,
            conv_descs_[i],
            algo, workspace, workspaceSizeInBytes,
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g], CUDNN_ADD_SAME_C,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)

    //calculate the mask 
    if(first_time_)
    {
      //LOG(INFO) << "First time forward"; 
      const Dtype* model_w_ = NULL;
    model_w_ = this->blobs_[0]->gpu_data();
    int dim1 = this->blobs_[0]->count(0, 1);
    int dim2 = this->blobs_[0]->count(1, 2);
    int kernel_num = this->blobs_[0]->count(2,4);
    Dtype* mask = mask_.mutable_cpu_data();
    for(int n_out = 0; n_out<dim1; n_out++)
    {
      for(int n_in = 0; n_in<dim2; n_in++)
      {
      Dtype flag = 0;
      caffe_gpu_asum(kernel_num,model_w_ + n_out*dim2*kernel_num + n_in*kernel_num, &flag);
      if(flag<0.0000000001) {
        //  LOG(INFO) << "zero: n_out:" << n_out << " n_in:" << n_in;
          caffe_set(kernel_num, (Dtype)(0.), mask + n_out*dim2*kernel_num + n_in*kernel_num);
      }
      else
      {
        caffe_set(kernel_num, (Dtype)(1.), mask + n_out*dim2*kernel_num + n_in*kernel_num);
      }
    }
    }
    first_time_ = false;  
    }
  



   sync_conv_groups_<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionMaskLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + weight_offset_ * g));
      }
      
      // My mask generator
      //Dtype* mask = mask_.gpu_data();
      caffe_gpu_mul(this->blobs_[0]->count(0, 4), weight_diff, mask_.gpu_data(),weight_diff);
     // int dim3 = this->blobs_[0]->count(2, 3);
     // int dim4 = this->blobs_[0]->count(3, 4);
     // int kernel_num = dim3*dim4;
      //Dtype zero_val = 0;
  //    const Dtype* weight_cpu = this->blobs_[0]->gpu_data();
    //  for(int n_out = 0; n_out<dim1; n_out++)
     // {
      //  for(int n_in = 0; n_in<dim2; n_in++)
      //  {
      //    Dtype flag = 0;
      //    caffe_gpu_asum(kernel_num,weight + n_out*dim2*kernel_num + n_in*kernel_num, &flag);
      //   if(flag<0.0000000001) {
      //      caffe_gpu_set(kernel_num, (Dtype)(0.), weight_diff + n_out*dim2*kernel_num + n_in*kernel_num);
      //    }
      //  }
     // }
      //cout << "end" << endl;


      // cout << "weight dim1: " << dim1 << endl;
      // cout << "weight dim2: " << dim2 << endl;
      // cout << "weight dim3: " << dim3 << endl;
      // cout << "weight dim4: " << dim4 << endl;

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
   sync_conv_groups_<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionMaskLayer);

}  // namespace caffe
#endif
