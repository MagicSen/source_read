#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w, const int ext_kernel_h, const int ext_kernel_w,
    const int stride_h, const int stride_w, const int kstride_h, const int kstride_w, const int pad_h, const int pad_w,
    Dtype* const top_data, int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    const int hend = min(hstart + ext_kernel_h, height);
    const int wend = min(wstart + ext_kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        if (bottom_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_slice[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w, const int ext_kernel_h, const int ext_kernel_w,
    const int stride_h, const int stride_w, const int kstride_h, const int kstride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + ext_kernel_h, height + pad_h);
    int wend = min(wstart + ext_kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += kstride_h) {
      for (int w = wstart; w < wend; w += kstride_w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
void PoolingSKLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, ext_kernel_h_, ext_kernel_w_,
        stride_h_, stride_w_, kstride_h_, kstride_w_,
        pad_h_, pad_w_, top_data,
        mask, top_mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, ext_kernel_h_, ext_kernel_w_,
        stride_h_, stride_w_, kstride_h_, kstride_w_,
        pad_h_, pad_w_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int* mask, const Dtype* const top_mask, const int num,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int ext_kernel_h, const int ext_kernel_w,
    const int stride_h, const int stride_w, const int kstride_h, const int kstride_w,
    const int pad_h, const int pad_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;

    int pooled_height_1 = pooled_height - 1;
    int pooled_width_1 = pooled_width - 1;
    int phstart =
        (h + pad_h < ext_kernel_h) ? h % kstride_h : (h + pad_h - ext_kernel_h) + 1;
    int phend = (h + pad_h >= pooled_height) ? pooled_height_1 - (pooled_height_1 - phstart) % kstride_h : h + pad_h;
    int pwstart =
        (w +pad_w < ext_kernel_w) ? w % kstride_w : (w + pad_w - ext_kernel_w) + 1;
    int pwend = (w + pad_w >= pooled_width) ? pooled_width_1 - (pooled_width_1 - pwstart) % kstride_w : w + pad_w;

    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    if (mask) {
      mask += offset;
      for (int ph = phstart; ph <= phend; ph += kstride_h) {
        for (int pw = pwstart; pw <= pwend; pw += kstride_w) {
          if (mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    } else {
      mask += offset;
      for (int ph = phstart; ph <= phend; ph += kstride_h) {
        for (int pw = pwstart; pw <= pwend; pw += kstride_w) {
          if (top_mask[ph * pooled_width + pw] == h * width + w) {
            gradient += top_diff[ph * pooled_width + pw];
          }
        }
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void PoolingSKLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  const int* mask = NULL;
  const Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->gpu_data();
    } else {
      mask = max_idx_.gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)

    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, top_mask, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_h_, kernel_w_, ext_kernel_h_, ext_kernel_w_,
        stride_h_, stride_w_, kstride_h_, kstride_w_,
        pad_h_, pad_w_,
        bottom_diff);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    NOT_IMPLEMENTED;
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PoolingSKLayer);


}  // namespace caffe
