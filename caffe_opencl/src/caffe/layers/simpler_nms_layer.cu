#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/fast_rcnn_layers.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA

extern "C" const char _cl_simpler_nms_layer_start;
extern "C" const char _cl_simpler_nms_layer_end;

namespace caffe {

template <typename Dtype>
void SimplerNMSLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
{
  std::vector<simpler_nms_proposal_t> sorted_proposals_confidence;

  if (this->device_->backend() == BACKEND_CUDA) {
    NOT_IMPLEMENTED;
  } else {
    const Dtype* bottom_cls_forward_prob = bottom[0]->gpu_data();
    const Dtype* bottom_deltas_pred = bottom[1]->gpu_data();
    int image_width = bottom[2]->cpu_data()[1];
    int image_height = bottom[2]->cpu_data()[0];
    int scaled_min_bbox_size = min_bbox_size_ * (int)bottom[2]->cpu_data()[2];
    int num_anchors = anchors_blob_.shape(0) * anchors_blob_.shape(1);
    int feature_map_width = bottom[0]->shape(3);
    int feature_map_height = bottom[0]->shape(2);
    int feature_map_size = feature_map_width * feature_map_height;
    cl_mem anchors_mem = (cl_mem) anchors_blob_.gpu_data();

    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->template program<Dtype>();

    int totalProps = feature_map_size * num_anchors;
    int propSize = 5 * sizeof(Dtype);
    Blob<Dtype> out_proposal_blob;
    out_proposal_blob.Reshape(1, 1, totalProps, propSize);
    cl_mem out_proposal_mem = (cl_mem) out_proposal_blob.mutable_gpu_data();

    viennacl::ocl::kernel &proposalForward = program.get_kernel(
        CL_KERNEL_SELECT("proposalForward"));
    proposalForward.global_work_size(0, feature_map_width);
    proposalForward.global_work_size(1, feature_map_height);
    proposalForward.global_work_size(2, num_anchors);

    proposalForward.local_work_size(0, feature_map_width);
    proposalForward.local_work_size(1, 1);
    proposalForward.local_work_size(2, 1);
    viennacl::ocl::enqueue(
        proposalForward(WrapHandle((cl_mem) bottom_deltas_pred, &ctx),
                        WrapHandle(anchors_mem, &ctx),
                        WrapHandle((cl_mem) bottom_cls_forward_prob, &ctx),
                        image_height, image_width, num_anchors,
                        feat_stride_, feature_map_size,
                        feature_map_width, scaled_min_bbox_size,
                        WrapHandle( out_proposal_mem, &ctx)),
                        ctx.get_queue());
    const Dtype *out_proposals = out_proposal_blob.cpu_data();
    for (unsigned int i = 0; i < totalProps; i++)
      if (out_proposals[i * 5 + 1] >= 0) {
        Dtype proposal_confidence = out_proposals[i * 5];
        simpler_nms_roi_t roi { out_proposals[i * 5 + 1],
                                out_proposals[i * 5 + 2],
                                out_proposals[i * 5 + 3],
                                out_proposals[i * 5 + 4] };
        simpler_nms_proposal_t proposal { roi, proposal_confidence, sorted_proposals_confidence.size() };
        sorted_proposals_confidence.push_back(proposal);
      }
  }

  sort_and_keep_at_most_top_n(sorted_proposals_confidence, pre_nms_topN_);

  auto res = simpler_nms_perform_nms(sorted_proposals_confidence, iou_threshold_, post_nms_topN_);
  size_t res_num_rois = res.size();

  Dtype* top_data = top[0]->mutable_cpu_data();
  for (size_t i = 0; i < res_num_rois; ++i)
  {
    top_data[5 * i + 0] = 0;    // roi_batch_ind, always zero on test time
    top_data[5 * i + 1] = res[i].x0;
    top_data[5 * i + 2] = res[i].y0;
    top_data[5 * i + 3] = res[i].x1;
    top_data[5 * i + 4] = res[i].y1;
  }

  top[0]->Reshape(vector<int>{ (int)res_num_rois, 5 });
}

template <typename Dtype>
void SimplerNMSLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
{
  // NOT_IMPLEMENTED;
}
INSTANTIATE_LAYER_GPU_FUNCS(SimplerNMSLayer);

}  // namespace caffe
#endif  // USE_GREENTEA
