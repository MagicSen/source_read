#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/upsample_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif


namespace caffe {

template <typename Dtype>
void UpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  int N = top[0]->shape(0);
  int C = top[0]->shape(1);
  int H = top[0]->shape(2);
  int W = top[0]->shape(3);

  const Dtype *input = bottom[0]->cpu_data();
  Dtype *output = top[0]->mutable_cpu_data();
  for (int n = 0; n < N; n++) {
    for (int c = 0; c < C; c++) {
      for (int h = 0; h < H; h++) {
        for (int w = 0; w < W; w++) {
          int nw = w/scale_;
          int nh = h/scale_;
          int out_idx = (((n * C + c) * H) + h) * W + w;
          int in_idx = (((n * C + c) * (H / scale_)) + nh) * (W / scale_) + nw;
          output[out_idx] = input[in_idx];
		  //std::cout << input[in_idx] << " ";
        }
		//std::cout << std::endl;
      }
    }
  }
}

/*
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w/scale_factor;
  z = z/scale_factor;
  d2 /= scale_factor;
  d3 /= scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ int translate_idx_inv(
    int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w*scale_factor+off_x;
  z = z*scale_factor+off_y;
  d2 *= scale_factor;
  d3 *= scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;
}

template <typename Dtype>
__global__ void upscale(const Dtype *input, Dtype *output,
        int no_elements, int scale_factor, int d1, int d2, int d3) {
  int ii = threadIdx.x + blockDim.x * blockIdx.x;
  if (ii >= no_elements) return;
  int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
  output[ii]=input[ipidx];
}

template <typename Dtype>
__global__ void downscale(Dtype *gradInput_data, const Dtype *gradOutput_data,
                          int no_elements, int scale_factor, int d1, int d2,
                          int d3) {
  int ii = threadIdx.x + blockDim.x * blockIdx.x;
  if (ii >= no_elements) return;
  for (int i = 0; i < scale_factor; i++) {
    for (int j = 0; j < scale_factor; j++) {
      int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
      gradInput_data[ii] += gradOutput_data[ipidx];
    }
  }
}


template <typename Dtype>
void UpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  int d1, d2, d3;

  d1 = top[0]->shape(1);
  d2 = top[0]->shape(2);
  d3 = top[0]->shape(3);

  int no_elements = top[0]->count();

  upscale<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(no_elements), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->gpu_data(),
      top[0]->mutable_gpu_data(), no_elements, scale_, d1, d2, d3);
}

template <typename Dtype>
void UpsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  int d1, d2, d3;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  d1 = bottom[0]->shape(1);
  d2 = bottom[0]->shape(2);
  d3 = bottom[0]->shape(3);
  int no_elements = bottom[0]->count();
  caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
  downscale<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(no_elements), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_diff, top[0]->gpu_diff(), no_elements, scale_, d1, d2, d3);
}
*/
INSTANTIATE_LAYER_GPU_FUNCS(UpsampleLayer);

}  // namespace caffe

